#include "hip/hip_runtime.h"
#include "global_include.h"
#include "cudaWrapperFunctions.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

using namespace std;

#ifdef LOW_PROFILE_PRINTING
#define print(input) std::cout << input << std::endl
#else
#define print(input) std::cout << "cuda functions: " << input << std::endl
#endif

bool allocateMemory(void** pointer, int size){
	hipError_t cudaStatus = hipMalloc(pointer, size);
	if (cudaStatus != hipSuccess)
	{
		print("allocate memory: " << hipGetErrorString(cudaStatus));
		return false;
	}
	return true;
}
bool freeMemory(void* pointer){
	hipError_t cudaStatus = hipFree(pointer);
	if (cudaStatus != hipSuccess){
		print("free memory: " << hipGetErrorString(cudaStatus));
		return false;
	}
	return true;
}

bool updateCudaDevice(void* pointerHost, void* pointerDevice, int size){
	hipError_t cudaStatus =
		// hipMemcpy(pointerHost, pointerDevice, size, hipMemcpyDeviceToHost);
		hipMemcpy(pointerDevice, pointerHost, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		print("update device: " << hipGetErrorString(cudaStatus));
		return false;
	}
	return true;
}
bool updateThisHost(void* pointerHost, void* pointerDevice, int size){
	
	hipError_t cudaStatus =
		hipMemcpy(pointerHost, pointerDevice, size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		print("update host: " << hipGetErrorString(cudaStatus));
		return false;
	}
	return true;
}


#define syncThreads __syncthreads()
#define syncDevice hipDeviceSynchronize()
#define THREAD_COUNT 128
#define LINEAR_GRID_SIZE(count) ((count - 1) / THREAD_COUNT + 1)

__device__ float squash(float input){
	return 2 / (1 + exp(-input)) - 1;
}

__device__ float invSquash(float input){
	return 2 / (1 - input * input);
}

__global__ void linearKernel(float* inputData, int inputLength,
	float* outputData, int outputLength, float* weights,
	float* biases){
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < outputLength){
		outputData[index] = biases[index];
		for (int i = 0; i < inputLength; i++){
			outputData[index] += inputData[i] * weights[i * outputLength + index];
		}
		outputData[index] = squash(outputData[index]);
	}
}

__global__ void linearBackKernel(float* inputData, int inputLength,
	float* outputData, int outputLength, float* weights,
	float* biases, float* inputBuffer, float epsilon){

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < inputLength){
		inputBuffer[index] = 0;
		for (int i = 0; i < outputLength; i++){
			inputBuffer[index] += outputData[i] * weights[index * outputLength + i] *
				invSquash(inputData[index]);
		}
	}
	syncThreads;
	if (index < outputLength){
		biases[index] += epsilon * outputData[index];
	}
	if (index < inputLength){
		for (int i = 0; i < outputLength; i++){
			weights[index * outputLength + i] += epsilon * outputData[i] * inputData[index];
		}
	}
	syncThreads;
	if (index < inputLength){
		for (int i = 0; i < outputLength; i++){
			inputData[index] = inputBuffer[index];
		}
	}
}

void runLinearKernel(dmem<float> from, dmem<float> to,
	dmem<float> weights, dmem<float> biases){

	dim3 grid(THREAD_COUNT);
	dim3 block(LINEAR_GRID_SIZE(from.size()));
	linearKernel<<<grid, block>>>(
		from, from.size(),
		to, to.size(),
		weights, biases);
	syncDevice;
}

void runLinearBackKernel(dmem<float> from, dmem<float> to,
	dmem<float> weights, dmem<float> biases, float epsilon){

	dmem<float> buffer(from.size());
	buffer.loadMemory();
	dim3 grid(THREAD_COUNT);
	dim3 block(LINEAR_GRID_SIZE(max(to.size(), from.size())));
	linearBackKernel<<<grid, block>>>(
		from, from.size(),
		to, to.size(),
		weights, biases,
		buffer, epsilon);
	syncDevice;
}
